
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__device__ float innerProduct(float* u, float* v, int uIndex, int vIndex) {

	return u[uIndex] * v[vIndex] + u[uIndex+1] * v[vIndex+1] + u[uIndex+2] * v[vIndex+2];
}

__device__ void crossProduct(float* u, float* v, int uIndex, int vIndex, float* result) {
	result[0] = v[vIndex + 1] * u[uIndex + 2] - v[vIndex + 2] * u[uIndex + 1]; //v X v2
        result[1] = v[vIndex + 2] * u[uIndex + 0] - v[vIndex + 0] * u[uIndex + 2];
        result[2] = v[vIndex + 0] * u[uIndex + 1] - v[vIndex + 1] * u[uIndex + 0];
}

__global__ void tracer(float* field, float* triangles) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int xIndex = index % 32 - 16;
	int yIndex = 16 - (int)(index / 32);


	float p[3] = {xIndex, yIndex, 1.0};
	float v[3] = {0.0, 0.0, -1.0};

	float v1[3], v2[3], cross[3], s[3]; //Triangle vectors
	float a, f, b, c, t;

	//printf("Triangle: [%f, %f, %f], [%f, %f, %f], [%f, %f, %f]\n", triangles[0], triangles[1], triangles[2], triangles[3], triangles[4], triangles[5], triangles[6], triangles[7], triangles[8]);
	//printf("Ray: [%f, %f, %f] -> [%f, %f, %f]\n", p[0], p[1], p[2], v[0], v[1], v[2]);


	for (int i = 0; i < 1; i += 9) {
		/*
		v1[0] = triangles[i + 0 + 0] - triangles[i + 3 + 0];
		v1[1] = triangles[i + 0 + 1] - triangles[i + 3 + 1];
		v1[2] = triangles[i + 0 + 2] - triangles[i + 3 + 2];
		v2[0] = triangles[i + 6 + 0] - triangles[i + 3 + 0];
		v2[1] = triangles[i + 6 + 1] - triangles[i + 3 + 1];
		v2[2] = triangles[i + 6 + 2] - triangles[i + 3 + 2];
		*/
		v1[0] = triangles[i + 0 + 0] - triangles[i + 3 + 0];
                v1[1] = triangles[i + 0 + 1] - triangles[i + 3 + 1];
                v1[2] = triangles[i + 0 + 2] - triangles[i + 3 + 2];
                v2[0] = triangles[i + 0 + 0] - triangles[i + 6 + 0];
                v2[1] = triangles[i + 0 + 1] - triangles[i + 6 + 1];
                v2[2] = triangles[i + 0 + 2] - triangles[i + 6 + 2];

		crossProduct(v, v2, 0, 0, cross); //v X v2

		a = innerProduct(v1, cross, 0, 0);
		
		if (a > -0.00001 && a < 0.00001) {
			field[index] = 0.1;
		} else {
			f = 1.0 / a;
			s[0] = triangles[i + 0 + 0] - p[0];
			s[1] = triangles[i + 0 + 1] - p[1];
			s[2] = triangles[i + 0 + 2] - p[2];

			b = f * innerProduct(s, cross, 0, 0);

			if (b < 0.0 || b > 1.0) {
				field[index] = 0.2;
			} else {

				crossProduct(s, v1, 0, 0, cross);
				c = f * innerProduct(v, cross, 0, 0);

				if (c < 0.0 || b + c > 1.0) {
					field[index] = 0.3;
				} else {

					t = f * innerProduct(v2, cross, 0, 0);
					if (t > 0.00001) {
						field[index] = 0.4;
					} else {
						field[index] = 1.1;
						printf("Ray: [%f, %f, %f] -> [%f, %f, %f]\n", p[0], p[1], p[2], v[0], v[1], v[2]);
					}
				}
			}
		}
	}

	
	//printf("[%d, %d] ", xIndex, yIndex);
	//field[index] = 1.0;
}
__global__ void hello() {
	printf("bb");
}

int main(void)
{
	int width = 32;
	int height = 32;

	const int trianglesCount = 1;
	/*float triangles[trianglesCount][3][3] = {
		{{-5.0, -5.0, 0.0}, {5.0, -5.0, 0.0}, {0.0, 5.0, 0.0}},
	};*/
	float triangles[9] = {
		-10.0, -10.0, 0.0, 
		10.0, -10.0, 0.0, 
		0.0, 10.0, 0.0
	};

	float near = 0.0;
	float far = 100.0;
	float left = width / -2.0;
	float right = width / 2.0;
	float top = height / 2.0;
	float bottom = height / -2.0;

	//To normalized device coordinates
	/*
	for (int i = 0; i < trianglesCount; i++) {
		for (int v = 0; v < 3; v++) {
			triangles[i][v][0] = triangles[i][v][0]	/ right;
			triangles[i][v][1] = triangles[i][v][1] / top;
			triangles[i][v][2] = triangles[i][v][2] / far;
		}
	}*/

	float result[width][height];
	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			result[i][j] = 0.0;
		}
	}

	int num_bytes = width * height * sizeof(float);
	int num_threads = width * height;
	int num_blocks = 1;

	float *host_array = 0;
	host_array = (float*)malloc(num_bytes);

	// cudaMalloc a device array
	float *device_array = 0;
	float *device_triangles = 0;
	hipMalloc((void**)&device_array, num_bytes);

	float triangles_num_bytes = 9 * sizeof(float);
	hipMalloc((void**)&device_triangles, triangles_num_bytes);

	hipMemcpy(device_triangles, triangles, triangles_num_bytes, hipMemcpyHostToDevice);


	//tracer<<<1,1>>>(device_array, device_triangles);
	tracer<<<num_blocks, num_threads>>>(device_array, device_triangles);
	hipDeviceSynchronize();

	// download and inspect the result on the host:
	hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);


	for (int i = 0; i < width; i++) {
                for (int j = 0; j < height; j++) {
			//std::cout << result[i][j] << ", ";
			std::cout << host_array[i * width + j] << " ";
                }
		std::cout << std::endl;
        }

	return 0;
}

